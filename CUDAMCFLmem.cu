/*	This file is part of CUDAMCFL.

    CUDAMCFL is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCFL is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCML_INC.  If not, see <http://www.gnu.org/licenses/>.*/

int CopyDeviceToHostMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{ //Copy data from Device to Host memory

	const int xy_size = sim->det.nx + sim->det.ny*sim->det.nx;
	const int num_x=(int)(4*(sim->esp)*(double)TAM_GRILLA);
	const int num_y=(int)(4*(sim->esp)*(double)TAM_GRILLA);
	const int num_z=(int)((sim->esp)*(double)TAM_GRILLA);
	//int banana_size = max_x*max_z+max_z;
	//const int fhd_size = num_x+num_x*(num_y+num_y*num_z);// x + HEIGHT* (y + WIDTH* z) //TODO: More efficient space usage
	const int fhd_size = num_x * num_y * num_z;
	//int xyz_size = sim->det.nx*sim->det.ny*sim->det.nz;

	//Copy Rd_xy, Tt_xy and A_xyz
	CUDA_SAFE_CALL( hipMemcpy(HostMem->Rd_xy,DeviceMem->Rd_xy,xy_size*sizeof(unsigned long long),hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(HostMem->Tt_xy,DeviceMem->Tt_xy,xy_size*sizeof(unsigned long long),hipMemcpyDeviceToHost) );


	CUDA_SAFE_CALL( hipMemcpy(HostMem->fhd,DeviceMem->fhd,fhd_size*sizeof(unsigned long long),hipMemcpyDeviceToHost) );

	//Also copy the state of the RNG's
	CUDA_SAFE_CALL( hipMemcpy(HostMem->x,DeviceMem->x,NUM_THREADS*sizeof(unsigned long long),hipMemcpyDeviceToHost) );

	//CUDA_SAFE_CALL( hipMemcpy(HostMem->last_func,DeviceMem->last_func,sizeof(unsigned short),hipMemcpyDeviceToHost) );

	return 0;
}


int InitDCMem(SimulationStruct* sim)
{
	unsigned int temp=0xFFFFFFFF;
	// Copy fhd flag
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(fhd_activated_dc),&(sim->fhd_activated),sizeof(unsigned int)) );

	// Copy det-data to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(det_dc),&(sim->det),sizeof(DetStruct)) );

	// Copy inclusion data to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(inclusion_dc),&(sim->inclusion),sizeof(IncStruct)) );

	// Copy number of layers to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(n_layers_dc),&(sim->n_layers),sizeof(unsigned int)));

	// Copy start_weight_dc to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(start_weight_dc),&(sim->start_weight),sizeof(unsigned int)));

	// Copy layer data to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(layers_dc),sim->layers,(sim->n_layers+2)*sizeof(LayerStruct)) );

	// Copy num_photons_dc to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(num_photons_dc),&(sim->number_of_photons),sizeof(unsigned long long)));

	// Copy x source position to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(xi_dc),&(sim->xi),sizeof(float)));
	// Copy y source position to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(yi_dc),&(sim->yi),sizeof(float)));
	// Copy z source position to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(zi_dc),&(sim->zi),sizeof(float)));
	// Copy source direction to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(dir_dc),&(sim->dir),sizeof(float)));
	// Copy esp to constant device memory
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(esp_dc),&(sim->esp),sizeof(float)));

	return 0;

}

int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{
	const int xy_size = sim->det.nx + sim->det.ny*sim->det.nx; //TODO: more efficient space usage

	const int num_x=(int)(4*(sim->esp)*(double)TAM_GRILLA);
	const int num_y=(int)(4*(sim->esp)*(double)TAM_GRILLA);
	const int num_z=(int)((sim->esp)*(double)TAM_GRILLA);
	//const int fhd_size = num_x+num_x*(num_y+num_y*num_z);////x + HEIGHT* (y + WIDTH* z)//TODO: more efficient space usage
	const int fhd_size = num_x * num_y * num_z;

	// Allocate p on the device
	CUDA_SAFE_CALL( hipMalloc((void**)&DeviceMem->p,NUM_THREADS*sizeof(PhotonStruct)) );

	// Allocate Rd_xy on CPU and GPU
	HostMem->Rd_xy = (unsigned long long*) malloc(xy_size*sizeof(unsigned long long));
	if(HostMem->Rd_xy==NULL){printf("Error allocating HostMem->Rd_xy"); exit (1);}
	CUDA_SAFE_CALL( hipMalloc((void**)&DeviceMem->Rd_xy,xy_size*sizeof(unsigned long long)) );
	CUDA_SAFE_CALL( hipMemset(DeviceMem->Rd_xy,0,xy_size*sizeof(unsigned long long)) );

	// Allocate Tt_xy on CPU and GPU
	HostMem->Tt_xy = (unsigned long long*) malloc(xy_size*sizeof(unsigned long long));
	if(HostMem->Tt_xy==NULL){printf("Error allocating HostMem->Tt_xy"); exit (1);}
	CUDA_SAFE_CALL( hipMalloc((void**)&DeviceMem->Tt_xy,xy_size*sizeof(unsigned long long)) );
	CUDA_SAFE_CALL( hipMemset(DeviceMem->Tt_xy,0,xy_size*sizeof(unsigned long long)) );

	// Allocate fhd on CPU and GPU
	HostMem->fhd = (unsigned long long*) malloc(fhd_size*sizeof(unsigned long long));
	if(HostMem->fhd==NULL){printf("Error allocating HostMem->fhd"); exit (1);}
	CUDA_SAFE_CALL( hipMalloc((void**)&DeviceMem->fhd,fhd_size*sizeof(unsigned long long)) );
	CUDA_SAFE_CALL( hipMemset(DeviceMem->fhd,0,fhd_size*sizeof(unsigned long long)) );

	// Allocate x and a on the device (For MWC RNG)
  CUDA_SAFE_CALL(hipMalloc((void**)&DeviceMem->x,NUM_THREADS*sizeof(unsigned long long)));
  CUDA_SAFE_CALL(hipMemcpy(DeviceMem->x,HostMem->x,NUM_THREADS*sizeof(unsigned long long),hipMemcpyHostToDevice));

  CUDA_SAFE_CALL(hipMalloc((void**)&DeviceMem->a,NUM_THREADS*sizeof(unsigned int)));
  CUDA_SAFE_CALL(hipMemcpy(DeviceMem->a,HostMem->a,NUM_THREADS*sizeof(unsigned int),hipMemcpyHostToDevice));


	// Allocate thread_active on the device and host
	HostMem->thread_active = (unsigned int*) malloc(NUM_THREADS*sizeof(unsigned int));
	if(HostMem->thread_active==NULL){printf("Error allocating HostMem->thread_active"); exit (1);}
	for(int i=0;i<NUM_THREADS;i++)HostMem->thread_active[i]=1u;

	CUDA_SAFE_CALL( hipMalloc((void**)&DeviceMem->thread_active,NUM_THREADS*sizeof(unsigned int)) );
	CUDA_SAFE_CALL( hipMemcpy(DeviceMem->thread_active,HostMem->thread_active,NUM_THREADS*sizeof(unsigned int),hipMemcpyHostToDevice));


	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = (unsigned long long*) malloc(sizeof(unsigned long long));
	if(HostMem->num_terminated_photons==NULL){printf("Error allocating HostMem->num_terminated_photons"); exit (1);}
	*HostMem->num_terminated_photons=0;

	CUDA_SAFE_CALL( hipMalloc((void**)&DeviceMem->num_terminated_photons,sizeof(unsigned long long)) );
	CUDA_SAFE_CALL( hipMemcpy(DeviceMem->num_terminated_photons,HostMem->num_terminated_photons,sizeof(unsigned long long),hipMemcpyHostToDevice));

	return 1;
}

void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{
	free(HostMem->Rd_xy);
	free(HostMem->Tt_xy);
	free(HostMem->fhd);
	free(HostMem->thread_active);
	free(HostMem->num_terminated_photons);

	hipFree(DeviceMem->p);
	hipFree(DeviceMem->Rd_xy);
	hipFree(DeviceMem->Tt_xy);
	hipFree(DeviceMem->fhd);
	hipFree(DeviceMem->x);
  hipFree(DeviceMem->a);
	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);

}

void FreeSimulationStruct(SimulationStruct* sim, int n_simulations)
{
	for(int i=0;i<n_simulations;i++)free(sim[i].layers);
	free(sim);
}
