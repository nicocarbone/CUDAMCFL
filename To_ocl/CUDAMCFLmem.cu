/*	This file is part of CUDAMCFL.

    CUDAMCFL is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCFL is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCFL.  If not, see <http://www.gnu.org/licenses/>.
*/

int CopyDeviceToHostMem(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{ // Copy data from Device to Host memory

	const int xy_size = sim->det.nx + sim->det.ny*sim->det.nx;
	const int num_x=(int)(4*(sim->esp)*(double)sim->grid_size);
	const int num_y=(int)(4*(sim->esp)*(double)sim->grid_size);
	const int num_z=(int)((sim->esp)*(double)sim->grid_size);
	const int fhd_size = num_x * num_y * num_z;

	const int num_x_tdet = sim->det.x_temp_numdets;
  const int num_y_tdet = sim->det.y_temp_numdets;
  const long num_tbins = sim->det.temp_bins;
  const long timegrid_size = num_x_tdet * num_y_tdet * num_tbins;

	// Copy Rd_xy, Tt_xy and A_xyz
	hipMemcpy(HostMem->Rd_xy,DeviceMem->Rd_xy,xy_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);
	hipMemcpy(HostMem->Tt_xy,DeviceMem->Tt_xy,xy_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);

	// Copy fhd
	hipMemcpy(HostMem->fhd,DeviceMem->fhd,fhd_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);

	// Copy time array
	hipMemcpy(HostMem->time_xyt,DeviceMem->time_xyt,timegrid_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);

	// Copy the state of the RNG's
	hipMemcpy(HostMem->x,DeviceMem->x,NUM_THREADS*sizeof(unsigned long long),hipMemcpyDeviceToHost);

	return 0;
}


int InitDCMem(SimulationStruct* sim)
{
	const int num_x=(int)(4*(sim->esp)*(double)sim->grid_size);
	const int num_y=(int)(4*(sim->esp)*(double)sim->grid_size);
	const int num_z=(int)((sim->esp)*(double)sim->grid_size);
	const int fhd_size = num_x * num_y * num_z;


	// Copy fhd flag
	hipMemcpy(fhd_activated_dc,&(sim->fhd_activated),sizeof(unsigned int), hipMemcpyHostToDevice);

	// Copy bulk method flag
	hipMemcpy(bulk_method_dc,&(sim->bulk_method),sizeof(unsigned int), hipMemcpyHostToDevice);

	// Copy time sim flag
	hipMemcpy(do_temp_sim_dc,&(sim->do_temp_sim),sizeof(unsigned int), hipMemcpyHostToDevice);


	// Copy det-data to constant device memory
	hipMemcpy(det_dc,&(sim->det),sizeof(DetStruct), hipMemcpyHostToDevice);

	// Copy inclusion data to constant device memory
	hipMemcpy(inclusion_dc,&(sim->inclusion),sizeof(IncStruct), hipMemcpyHostToDevice);

	// Copy number of layers to constant device memory
	hipMemcpy(n_layers_dc,&(sim->n_layers),sizeof(unsigned int), hipMemcpyHostToDevice);

	// Copy number of bulk descriptors to constant device memory
	hipMemcpy(n_bulks_dc,&(sim->n_bulks),sizeof(unsigned int), hipMemcpyHostToDevice);

	// Copy start_weight_dc to constant device memory
	hipMemcpy(start_weight_dc,&(sim->start_weight),sizeof(unsigned int), hipMemcpyHostToDevice);

	// Copy grid_size_dc to constant device memory
	hipMemcpy(grid_size_dc,&(sim->grid_size),sizeof(unsigned int), hipMemcpyHostToDevice);

	// Copy layer data to constant device memory
	hipMemcpy(layers_dc,sim->layers,(sim->n_layers+2)*sizeof(LayerStruct), hipMemcpyHostToDevice);

	// Copy bulk data to constant device memory
	hipMemcpy(bulks_dc,sim->bulks,(sim->n_bulks+2)*sizeof(BulkStruct), hipMemcpyHostToDevice);

	// Copy num_photons_dc to constant device memory
	hipMemcpy(num_photons_dc,&(sim->number_of_photons),sizeof(unsigned long long), hipMemcpyHostToDevice);

	// Copy x source position to constant device memory
	hipMemcpy(xi_dc,&(sim->xi),sizeof(float), hipMemcpyHostToDevice);

	// Copy y source position to constant device memory
	hipMemcpy(yi_dc,&(sim->yi),sizeof(float), hipMemcpyHostToDevice);

	// Copy z source position to constant device memory
	hipMemcpy(zi_dc,&(sim->zi),sizeof(float), hipMemcpyHostToDevice);

	// Copy source direction to constant device memory
	hipMemcpy(dir_dc,&(sim->dir),sizeof(float), hipMemcpyHostToDevice);

	// Copy esp to constant device memory
	hipMemcpy(esp_dc,&(sim->esp),sizeof(float), hipMemcpyHostToDevice);

	return 0;

}

int InitMemStructs(MemStruct* HostMem, MemStruct* DeviceMem, SimulationStruct* sim)
{
	const int xy_size = sim->det.nx + sim->det.ny*sim->det.nx; //TODO: more efficient space usage

	const int num_x=(int)(4*(sim->esp)*(double)sim->grid_size);
	const int num_y=(int)(4*(sim->esp)*(double)sim->grid_size);
	const int num_z=(int)((sim->esp)*(double)sim->grid_size);
	const int fhd_size = num_x * num_y * num_z;

	const int num_x_tdet = sim->det.x_temp_numdets;
  const int num_y_tdet = sim->det.y_temp_numdets;
  const long num_tbins = sim->det.temp_bins;
  const long timegrid_size = num_x_tdet * num_y_tdet * num_tbins;

	// Allocate p on the device
	hipMalloc((void**)&DeviceMem->p,NUM_THREADS*sizeof(PhotonStruct));

	// Allocate Rd_xy on CPU and GPU
	HostMem->Rd_xy = (unsigned long long*) malloc(xy_size*sizeof(unsigned long long));
	if(HostMem->Rd_xy==NULL){printf("Error allocating HostMem->Rd_xy"); exit (1);}
	hipMalloc((void**)&DeviceMem->Rd_xy,xy_size*sizeof(unsigned long long));
	hipMemset(DeviceMem->Rd_xy,0,xy_size*sizeof(unsigned long long));

	// Allocate Tt_xy on CPU and GPU
	HostMem->Tt_xy = (unsigned long long*) malloc(xy_size*sizeof(unsigned long long));
	if(HostMem->Tt_xy==NULL){printf("Error allocating HostMem->Tt_xy"); exit (1);}
	hipMalloc((void**)&DeviceMem->Tt_xy,xy_size*sizeof(unsigned long long));
	hipMemset(DeviceMem->Tt_xy,0,xy_size*sizeof(unsigned long long));

	// Allocate fhd on CPU and GPU
	HostMem->fhd = (unsigned long long*) malloc(fhd_size*sizeof(unsigned long long));
	if(HostMem->fhd==NULL){printf("Error allocating HostMem->fhd"); exit (1);}
	hipMalloc((void**)&DeviceMem->fhd,fhd_size*sizeof(unsigned long long));
	hipMemset(DeviceMem->fhd,0,fhd_size*sizeof(unsigned long long));

	// Allocate timegrid on CPU and GPU
	HostMem->time_xyt = (unsigned long long*) malloc(timegrid_size*sizeof(unsigned long long));
	if(HostMem->time_xyt==NULL){printf("Error allocating HostMem->time_xyt"); exit (1);}
	hipMalloc((void**)&DeviceMem->time_xyt,timegrid_size*sizeof(unsigned long long));
	hipMemset(DeviceMem->time_xyt,0,timegrid_size*sizeof(unsigned long long));

	// Allocate x time detectors
	HostMem->tdet_pos_x = (float*) malloc(num_x_tdet*sizeof(float));
	if(HostMem->tdet_pos_x==NULL){printf("Error allocating HostMem->tdet_pos_x"); exit (1);}
	hipMalloc((void**)&DeviceMem->tdet_pos_x,num_x_tdet*sizeof(float));
	hipMemset(DeviceMem->tdet_pos_x,0,num_x_tdet*sizeof(float));

	// Allocate y time detectors
	HostMem->tdet_pos_y = (float*) malloc(num_y_tdet*sizeof(float));
	if(HostMem->tdet_pos_y==NULL){printf("Error allocating HostMem->tdet_pos_y"); exit (1);}
	hipMalloc((void**)&DeviceMem->tdet_pos_y,num_y_tdet*sizeof(float));
	hipMemset(DeviceMem->tdet_pos_y,0,num_y_tdet*sizeof(float));

	// Allocate x and a on the device (For MWC RNG)
  hipMalloc((void**)&DeviceMem->x,NUM_THREADS*sizeof(unsigned long long));
  hipMemcpy(DeviceMem->x,HostMem->x,NUM_THREADS*sizeof(unsigned long long),hipMemcpyHostToDevice);

  hipMalloc((void**)&DeviceMem->a,NUM_THREADS*sizeof(unsigned int));
  hipMemcpy(DeviceMem->a,HostMem->a,NUM_THREADS*sizeof(unsigned int),hipMemcpyHostToDevice);

	// Allocate bulk_info 3D matrix and copy to device memory
	hipMalloc((void**)&DeviceMem->bulk_info,fhd_size*sizeof(short));
	hipMemcpy(DeviceMem->bulk_info,sim->bulk_info,fhd_size*sizeof(short), hipMemcpyHostToDevice );


	// Allocate thread_active on the device and host
	HostMem->thread_active = (unsigned int*) malloc(NUM_THREADS*sizeof(unsigned int));
	if(HostMem->thread_active==NULL){printf("Error allocating HostMem->thread_active"); exit (1);}
	for(int i=0;i<NUM_THREADS;i++)HostMem->thread_active[i]=1u;
	hipMalloc((void**)&DeviceMem->thread_active,NUM_THREADS*sizeof(unsigned int));
	hipMemcpy(DeviceMem->thread_active,HostMem->thread_active,NUM_THREADS*sizeof(unsigned int),hipMemcpyHostToDevice);

	//Allocate num_launched_photons on the device and host
	HostMem->num_terminated_photons = (unsigned long long*) malloc(sizeof(unsigned long long));
	if(HostMem->num_terminated_photons==NULL){printf("Error allocating HostMem->num_terminated_photons"); exit (1);}
	*HostMem->num_terminated_photons=0;
	hipMalloc((void**)&DeviceMem->num_terminated_photons,sizeof(unsigned long long));
	hipMemcpy(DeviceMem->num_terminated_photons,HostMem->num_terminated_photons,sizeof(unsigned long long),hipMemcpyHostToDevice);

	return 1;
}

void FreeMemStructs(MemStruct* HostMem, MemStruct* DeviceMem)
{
	free(HostMem->Rd_xy);
	free(HostMem->Tt_xy);
	free(HostMem->time_xyt);
	free(HostMem->tdet_pos_x);
	free(HostMem->tdet_pos_y);
	free(HostMem->fhd);
	free(HostMem->thread_active);
	free(HostMem->num_terminated_photons);

	hipFree(DeviceMem->p);
	hipFree(DeviceMem->Rd_xy);
	hipFree(DeviceMem->Tt_xy);
	hipFree(DeviceMem->time_xyt);
	hipFree(DeviceMem->tdet_pos_x);
	hipFree(DeviceMem->tdet_pos_y);
	hipFree(DeviceMem->fhd);
	hipFree(DeviceMem->bulk_info);
	hipFree(DeviceMem->x);
  hipFree(DeviceMem->a);
	hipFree(DeviceMem->thread_active);
	hipFree(DeviceMem->num_terminated_photons);

}

void FreeSimulationStruct(SimulationStruct* sim, int n_simulations)
{
	for(int i=0;i<n_simulations;i++)free(sim[i].layers);
	free(sim);
}
