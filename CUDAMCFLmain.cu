#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////
//
// Monte Carlo simulation software for light propagation in fluorescent turbid media,
// accelerated by GPU (graphic processing unit).
// The code is based on previous work by Alerstam et al and Wang et al,
// with the addition of a voxelized medium without symmetries and with an
// inhomogeneous distribution of absorbers and fluorescent marker
//
///////////////////////////////////////////////////////////////

/*	This file is part of CUDAMCFL.

    CUDAMCFL is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCFL is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCFL.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "CUDAMCFL.h"
#include "cutil.h"
//#include <float.h> //for FLT_MAX
#include <limits.h>
#include <stdio.h>
#include "hip/hip_runtime_api.h"

__device__ __constant__ unsigned long long num_photons_dc[1];
__device__ __constant__ unsigned int n_layers_dc[1];
__device__ __constant__ unsigned int n_bulks_dc[1];
__device__ __constant__ unsigned int start_weight_dc[1];
__device__ __constant__ LayerStruct layers_dc[MAX_LAYERS];
__device__ __constant__ BulkStruct bulks_dc[MAX_LAYERS];
__device__ __constant__ DetStruct det_dc[1];
__device__ __constant__ IncStruct inclusion_dc[1];
__device__ __constant__ unsigned int ignoreAdetection_dc[1];
__device__ __constant__ unsigned int fhd_activated_dc[1];
__device__ __constant__ unsigned int do_temp_sim_dc[1];
__device__ __constant__ unsigned int bulk_method_dc[1];
__device__ __constant__ float xi_dc[1];
__device__ __constant__ float yi_dc[1];
__device__ __constant__ float zi_dc[1];
__device__ __constant__ float dir_dc[1];
__device__ __constant__ float esp_dc[1];
__device__ __constant__ unsigned int grid_size_dc[1];
__device__ __constant__ unsigned int max_temp_dc[1];

#include "CUDAMCFLio.cu"
#include "CUDAMCFLmem.cu"
#include "CUDAMCFLrng.cu"
#include "CUDAMCFLtransport.cu"

// wrapper for device code - FHD Simulation
unsigned long long DoOneSimulation(SimulationStruct *simulation, unsigned long long *x,
                     unsigned int *a, double *tempfhd, double* temptdist) {
  MemStruct DeviceMem;
  MemStruct HostMem;
  unsigned int threads_active_total = 1;
  unsigned int i, ii;

  // Output matrix size
  const int num_x = (int)(4 * (simulation->esp) * (float)simulation->grid_size);
  const int num_y = (int)(4 * (simulation->esp) * (float)simulation->grid_size);
  const int num_z = (int)((simulation->esp) * (float)simulation->grid_size);
  const int fhd_size = num_x * num_y * num_z;

  // Output temporal detectors
  const int num_x_tdet = simulations[0].det.x_temp_numdets;
  const int num_y_tdet = simulations[0].det.y_temp_numdets;
  const long num_tbins = simulations[0].det.temp_bins;
  const long timegrid_size = num_x_tdet * num_y_tdet * num_tbins;

  hipError_t cudastat;
  clock_t time1, time2;

  // Start the clock
  time1 = clock();

  // x and a are already initialised in memory
  HostMem.x = x;
  HostMem.a = a;

  InitMemStructs(&HostMem, &DeviceMem, simulation);

  InitDCMem(simulation);

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(NUM_BLOCKS);
  int blockSize;   // The launch configurator returned block size
  int minGridSize; // The minimum grid size needed to achieve the
                   // maximum occupancy for a full device launch
  //int gridSize;    // The actual grid size needed, based on input size

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
                                      MCd3D, 0, 0);
  printf ("Grid size: %i, Block size: %i \n\n", minGridSize, blockSize);


  LaunchPhoton_Global<<<dimGrid, dimBlock>>>(DeviceMem);
  //LaunchPhoton_Global<<<minGridSize, blockSize>>>(DeviceMem);
  CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
  cudastat = hipGetLastError();           // Check if there was an error
  if (cudastat)
    printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

  i = 0;
  while (threads_active_total > 0) {
    i++;
    // run the kernel
    if (simulation->bulk_method == 1){
      MCd<<<dimGrid, dimBlock>>>(DeviceMem);
      //MCd<<<minGridSize, blockSize>>>(DeviceMem);

    }
    else if (simulation->bulk_method == 2) {

      MCd3D<<<dimGrid, dimBlock>>>(DeviceMem);
      //MCd3D<<<minGridSize, blockSize>>>(DeviceMem);
    }

    CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
    cudastat = hipGetLastError();           // Check if there was an error
    if (cudastat)
      printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

    // Copy thread_active from device to host
    CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active,
                              NUM_THREADS * sizeof(unsigned int),
                              hipMemcpyDeviceToHost));
    threads_active_total = 0;
    for (ii = 0; ii < NUM_THREADS; ii++)
      threads_active_total += HostMem.thread_active[ii];

    CUDA_SAFE_CALL(hipMemcpy(HostMem.num_terminated_photons,
                              DeviceMem.num_terminated_photons,
                              sizeof(unsigned long long), hipMemcpyDeviceToHost));
    if (i == 100)
      printf("Estimated PHD simulation time: %.0f secs.\n\n",
             (double)(clock() - time1) / CLOCKS_PER_SEC *
                 (double)(simulation->number_of_photons /
                          *HostMem.num_terminated_photons));
//    if (fmod(i, 200u) == 0) printf("."); fflush(stdout);
    if (i % 100 == 0) printf("."); fflush(stdout);
    if (i % 2000 == 0)
      printf("\nRun %u, %llu photons simulated\n", i,
             *HostMem.num_terminated_photons);
  }

  CopyDeviceToHostMem(&HostMem, &DeviceMem, simulation);

  time2 = clock();

  printf("\nSimulation time: %.2f sec\n\n",
         (double)(time2 - time1) / CLOCKS_PER_SEC);

  printf("Writing excitation results...\n");
  Write_Simulation_Results(&HostMem, simulation, time2-time1);
  printf("PHD Simulation done!\n");

  unsigned long long photons_finished = *HostMem.num_terminated_photons;

  // Normalize and write output matrix
  for (int xyz = 0; xyz < fhd_size; xyz++) {
    tempfhd[xyz] = ((double)HostMem.fhd[xyz]/(0xFFFFFFFF*photons_finished));
  }

  // Normalize and write output matrix
  //for (int xyz = 0; xyz < timegrid_size; xyz++) {
  //  tgrid[xyz] = ((double)HostMem.time_xyt[xyz]/(0xFFFFFFFF*photons_finished));
  //}

  printf ("Photons simulated: %llu\n\n", photons_finished);
  FreeMemStructs(&HostMem, &DeviceMem);
  return photons_finished;
}

// wrapper for device code - fluorescence Simulation
unsigned long long DoOneSimulationFl(SimulationStruct *simulation, unsigned long long *x,
                       unsigned int *a, unsigned long long *tempvoxelR, unsigned long long *tempvoxelT) {
  MemStruct DeviceMem;
  MemStruct HostMem;
  unsigned int threads_active_total = 1;
  unsigned int i, ii;

  // Size of output matrix
  const int nx2 = simulation->det.nx;
  const int ny2 = simulation->det.ny;
  const int xy_size = nx2 + ny2 * nx2;

  hipError_t cudastat;

  // x and a are already initialised in memory
  HostMem.x = x;
  HostMem.a = a;

  InitMemStructs(&HostMem, &DeviceMem, simulation);

  InitDCMem(simulation);

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(NUM_BLOCKS);
  int blockSize;   // The launch configurator returned block size
  int minGridSize; // The minimum grid size needed to achieve the
                   // maximum occupancy for a full device launch
  //int gridSize;    // The actual grid size needed, based on input size T

  hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
                                      MCd, 0, 0); //TODO
  //printf ("Grid size: %i, Block size: %i \n\n", minGridSize, blockSize);

  LaunchPhoton_Global<<<dimGrid, dimBlock>>>(DeviceMem);
  //LaunchPhoton_Global<<<minGridSize, blockSize>>>(DeviceMem);

  CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
  cudastat = hipGetLastError();           // Check if there was an error
  if (cudastat)
    printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

  i = 0;
  while (threads_active_total > 0) {
    i++;
    // run the kernel
    if (simulation->bulk_method == 1){
      MCd<<<dimGrid, dimBlock>>>(DeviceMem);
      //MCd<<<minGridSize, blockSize>>>(DeviceMem);

    }
    else if (simulation->bulk_method == 2) {
      MCd3D<<<dimGrid, dimBlock>>>(DeviceMem);
      //MCd3D<<<minGridSize, blockSize>>>(DeviceMem);
    }

    CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
    cudastat = hipGetLastError();           // Check if there was an error
    if (cudastat)
      printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

    // Copy thread_active from device to host
    CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active,
                              NUM_THREADS * sizeof(unsigned int),
                              hipMemcpyDeviceToHost));
    threads_active_total = 0;
    for (ii = 0; ii < NUM_THREADS; ii++)
      threads_active_total += HostMem.thread_active[ii];

    CUDA_SAFE_CALL(hipMemcpy(HostMem.num_terminated_photons,
                              DeviceMem.num_terminated_photons,
                              sizeof(unsigned long long), hipMemcpyDeviceToHost));

    if (i > 10000) {
      // If we are still running after 10000 steps, something definetly went wrong.
      printf("\nWARNING: Breaking out of loop...\n");
      return 0;
    }
  }


  CopyDeviceToHostMem(&HostMem, &DeviceMem, simulation);

  for (int ijk = 0; ijk < xy_size; ijk++) {
    // Reflection
    tempvoxelR[ijk] = HostMem.Rd_xy[ijk];
    // Transmission
    tempvoxelT[ijk] = HostMem.Tt_xy[ijk];
  }

  unsigned long long photons_finished = *HostMem.num_terminated_photons;
  FreeMemStructs(&HostMem, &DeviceMem);
  return photons_finished;
}

int main(int argc, char *argv[]) {

  printf ("\nCUDAMCFL. Compilation date: %s, %s. \n", __DATE__, __TIME__);

  clock_t time0 = clock();
  SimulationStruct *simulations;
  int n_simulations;
  unsigned long long seed =
      (unsigned long long)time(NULL); // Default, use time(NULL) as seed
  int ignoreAdetection = 0;
  char *filename;
  char *filenameflR;
  char *filenameflT;
  unsigned long fhd_sim_photons;

  if (argc < 2) {
    printf("Not enough input arguments!\n");
    return 1;
  } else {
    filename = argv[1];
  }

  printf("\nExecuting %s... \n", filename);
  printf("____________________________________________________________________\n\n");

  if (interpret_arg(argc, argv, &seed, &ignoreAdetection))
    return 1;

  n_simulations =
      read_simulation_data(filename, &simulations, ignoreAdetection);

  if (n_simulations == 0) {
    printf("Something wrong with read_simulation_data!\n");
    return 1;
  } else {
    printf("\nRead %d simulations\n\n", n_simulations);
  }


  // Allocate memory for RNG's
  unsigned long long x[NUM_THREADS];
  unsigned int a[NUM_THREADS];

  // Init RNG's
  if (init_RNG(x, a, NUM_THREADS, "safeprimes_base32.txt", seed))
    return 1;

  // Store in local variables the number of voxels in each direction
  const int num_x = (int)(4 * (simulations[0].esp) * simulations[0].grid_size);
  const int num_y = (int)(4 * (simulations[0].esp) * simulations[0].grid_size);
  const int num_z = (int)((simulations[0].esp) * simulations[0].grid_size);

  const int fhd_size = num_x * num_y * num_z; //x + HEIGHT* (y + WIDTH* z)


  // Store in local variables the number of time detectors

  const int num_x_tdet = simulations[0].det.x_temp_numdets;
  const int num_y_tdet = simulations[0].det.y_temp_numdets;
  const long num_tbins = simulations[0].det.temp_bins;

  const long timegrid_size = num_x_tdet * num_y_tdet * num_tbins;

  // FHD simulation
  // Run a simulation

  const unsigned long long number_phd_photons = simulations[0].number_of_photons;

  printf("Running PHD simulation...\n");

  double *Fx;
  Fx = (double *)malloc((fhd_size) * sizeof(double));

  //double *Tgrid;
  //Tgrid = (double *)malloc((timegrid_size) * sizeof(double));

  fhd_sim_photons = DoOneSimulation(&simulations[0], x, a, Fx);

  if(simulations[0].fhd_activated==1){
    // Outputting FHD files for debug
    printf("Writing PHD files...\n"); // TODO

    // ASCII file

    FILE *fhd3DaFile_out;
    char filenamefl3da[STR_LEN];
	   for (int ic=0; ic<STR_LEN; ic++) filenamefl3da[ic] = simulations[0].outp_filename[ic];
     strcat(filenamefl3da, "_PHD-Ascii.dat");

     fhd3DaFile_out = fopen(filenamefl3da, "w");
     if (fhd3DaFile_out == NULL) {
       perror("Error opening output file");
       return 0;
     }

     fprintf(fhd3DaFile_out, "%llu\t%llu\t%llu\n", num_x,num_y,num_z);

     for (int xyz = 0; xyz < fhd_size; xyz++) {
       fprintf(fhd3DaFile_out, "%.10E\n", Fx[xyz]);
     }

     fclose(fhd3DaFile_out);
   }

  /*
  // Binary file
  FILE *fhd3DbFile_out;
  char filenamefl3db[STR_LEN];
	for (int ic=0; ic<STR_LEN; ic++) filenamefl3db[ic] = simulations[0].outp_filename[ic];
  strcat(filenamefl3db, "_FHD-Binary.dat");

  fhd3DbFile_out = fopen(filenamefl3db, "wb");
  if (fhd3DbFile_out == NULL) {
    perror("Error opening output file");
    return 0;
  }

  */

  // Fluorescence simulation

  // Initialize GPU RNG
  seed = (unsigned long long)time(NULL); // Default, use time(NULL) as seed
  if (init_RNG(x, a, NUM_THREADS, "safeprimes_base32.txt", seed))
    return 1;

  unsigned long fluor_sim_photons = 0; //Nro of simulated fluorescence photons

  if (simulations[0].do_fl_sim != 0){
    printf("Flourescence simulation... \n");
    int count_failed = 0;
    // Store in local variables the number of pixels and calculate image size
    const int nx2 = simulations[0].det.nx;
    const int ny2 = simulations[0].det.ny;
    const int xy_size = nx2 + ny2 * nx2;

    // Pixel size for Normalization
    const double dx = simulations[0].det.dx;
    const double dy = simulations[0].det.dy;

    // Initialize arrays
    double *Fl_HetR, *Fl_HetT;          // Final fluorescence image
    long voxel_finished = 0; // Nro of voxel simulated
    long voxel_inside = 0;   // Nro of voxel simulated inside inclusion
    long voxel_outside = 0;  // Nro of voxel simulated outside inclusion
    //const long for_size = num_x * num_y * num_z; // Total number of voxels to be simulated
    float xi, yi, zi;          // Temporal variable to store the voxel coordinates
    double voxelw; // Temporal variable to store the voxel scale factor
    clock_t time1,
        time2, time3; // Variable to store the timestamps used for run time stimation

    // Allocate and initialize to zero image matrix
    Fl_HetR = (double *)malloc(xy_size * sizeof(double));
    for (int ijk = 0; ijk < xy_size; ijk++) {
      Fl_HetR[ijk] = 0.0;
    }
    // Allocate and initialize to zero image matrix
    Fl_HetT = (double *)malloc(xy_size * sizeof(double));
    for (int ijk = 0; ijk < xy_size; ijk++) {
      Fl_HetT[ijk] = 0.0;
    }

    // Simulations parameters
    for (int n = 0; n < simulations[0].n_layers + 2;
        n++) { // Set mua to fluorescence value for every layer
        simulations[0].layers[n].mua = simulations[0].layers[n].muaf;
    }

    for (int n = 0; n < simulations[0].n_bulks + 2;
        n++) { // Set mua to fluorescence value for every layer
        simulations[0].bulks[n].mua = simulations[0].bulks[n].muaf;
    }

    simulations[0].number_of_photons = (unsigned long long)simulations[0].number_of_photons_per_voxel; // Number of photons per voxel
    simulations[0].dir = 0.0f;        // Isotropic source
    simulations[0].fhd_activated = 0; // Don't accumulate fhd

    printf("Total fotons to be simulated: %lli over %li voxels\n",
          simulations[0].number_of_photons * fhd_size, fhd_size);

    // Loop through the voxels
    for (int ix = 0; ix < num_x; ix++) {
      for (int iy = 0; iy < num_y; iy++) {
        for (int iz = 0; iz < num_z; iz++) {
          if (ix == 0 && iy == 0 && iz == 0)
            time1 = clock(); // For the first xyz voxel, take first timestamp

          int index = ix + num_x * (iy + iz * num_y);
          short bulkdescriptor = simulations[0].bulk_info[index];

          // Set source position
          xi = ((float) ix / simulations[0].grid_size) - 2* simulations[0].esp;
          yi = ((float) iy / simulations[0].grid_size) - 2* simulations[0].esp;
          zi = ((float) iz / simulations[0].grid_size);

          simulations[0].xi = xi;
          simulations[0].yi = yi;
          simulations[0].zi = zi;

          // Locate layer of voxel (we need it to retrieve apropiate albedo)
          int found = 0;
          int nl = 1;
          while (nl < simulations[0].n_layers + 2 && found != 1) {
            if (zi < simulations[0].layers[nl].z_max &&
                zi >= simulations[0].layers[nl].z_min) {
              found = 1;
            } else
              nl++;
          }

          // Do the voxel simulation
          unsigned long long *tempretR;
          tempretR =
              (unsigned long long *)malloc(xy_size * sizeof(unsigned long long));
          unsigned long long *tempretT;
          tempretT =
              (unsigned long long *)malloc(xy_size * sizeof(unsigned long long));

          unsigned long long voxel_status;

          // Check if inside inclusion and calculate scale value accordingly
          if (simulations[0].bulk_method == 1){
            if ((xi - simulations[0].inclusion.x) *
                        (xi - simulations[0].inclusion.x) +
                  (yi - simulations[0].inclusion.y) *
                      (yi - simulations[0].inclusion.y) +
                  (zi - simulations[0].inclusion.z) *
                      (zi - simulations[0].inclusion.z) <
                simulations[0].inclusion.r * simulations[0].inclusion.r) {
            // voxel inside inclusion
              if (simulations[0].inclusion.albedof<0){
                voxel_status = DoOneSimulationFl(&simulations[0], x, a, tempretR, tempretT);
                voxelw = ((double)simulations[0].inclusion.eY *
                    (double)(1 - simulations[0].inclusion.albedof) *
                    Fx[index]) /
                    (double)(voxel_status * 0xFFFFFFFF);
                  }
              else {
                voxelw=0;
                voxel_status=1;
              }
              voxel_inside++;
            }
            else {
            // voxel ouside inclusion
              if (simulations[0].layers[nl].albedof<0){
                voxel_status = DoOneSimulationFl(&simulations[0], x, a, tempretR, tempretT);
                voxelw = ((double)simulations[0].layers[nl].eY *
                    (double)(1 - simulations[0].layers[nl].albedof) *
                    Fx[index]) /
                    (double)(voxel_status * 0xFFFFFFFF);
                }
              else {
                voxelw=0;
                voxel_status=1;
              }
              voxel_outside++;
            }
          }

          if (simulations[0].bulk_method == 2){
            if (simulations[0].bulks[bulkdescriptor].albedof<1){
              voxel_status = DoOneSimulationFl(&simulations[0], x, a, tempretR, tempretT);
              voxelw = ((double)simulations[0].bulks[bulkdescriptor].eY *
                    (double)(1 - simulations[0].bulks[bulkdescriptor].albedof) *
                    Fx[index]) /
                    (double)(voxel_status * 0xFFFFFFFF);
              }
              else {
                voxelw=0;
                voxel_status=1;
              }
            }

          if (voxel_status == 0) {
            printf("Voxel %f, %f, %f failed.\n", xi,yi,zi);
            count_failed += 1;
            }

          fluor_sim_photons += voxel_status;

          // Accumulate image
          for (int ij = 0; ij < xy_size; ij++) {
            double tempvwR = voxelw * (double)tempretR[ij];
            double tempvwT = voxelw * (double)tempretT[ij];
            if (Fl_HetR[ij] + tempvwR < DBL_MAX) Fl_HetR[ij] += tempvwR/(dx*dy);
            if (Fl_HetT[ij] + tempvwT < DBL_MAX) Fl_HetT[ij] += tempvwT/(dx*dy);
					}
          voxel_finished++;

          free(tempretR);
          free(tempretT);

          if (voxel_finished % 200 == 0) printf("."); fflush(stdout);
          if (voxel_finished % 10000 == 0)
            printf("\n%li of %li voxels finished\n", voxel_finished, fhd_size);
          if (voxel_finished == 199) { // Second timestamp after 99 voxels run (so
                                    // it displays before the first progression
                                    // report)
            printf("Estimated fluorescence simulation time: %.0f sec\n\n",
                 (double)(clock() - time1) * fhd_size / CLOCKS_PER_SEC / 199);
          }
        }
      }
    }

    printf("\n\nFlourescence simulation finished!\n");

    if (simulations[0].bulk_method == 1){
      printf("Voxels inside inclusion: %li\n", voxel_inside);
      printf("Voxels outside inclusion: %li\n", voxel_outside);
      printf("Voxels failed: %i\n", count_failed);
    }

    printf("Writing results files...\n"); // TODO
    FILE *fhdRFile_out;
    char filenameflR[STR_LEN];
  	for (int ic=0; ic<STR_LEN; ic++) filenameflR[ic] = simulations[0].outp_filename[ic];
    strcat(filenameflR, "_FlR.dat");

    fhdRFile_out = fopen(filenameflR, "w");
    if (fhdRFile_out == NULL) {
      perror("Error opening output file");
      return 0;
    }

    for (int y = 0; y < ny2; y++) {
      for (int x = 0; x < nx2; x++) {
        fprintf(fhdRFile_out, " %E ", Fl_HetR[y * nx2 + x]);
      }
      fprintf(fhdRFile_out, " \n ");
    }

    fclose(fhdRFile_out);
    // Free memory
    free(Fl_HetR);

    FILE *fhdTFile_out;
    char filenameflT[STR_LEN];
  	for (int ic=0; ic<STR_LEN; ic++) filenameflT[ic] = simulations[0].outp_filename[ic];
    strcat(filenameflT, "_FlT.dat");

    fhdTFile_out = fopen(filenameflT, "w");
    if (fhdTFile_out == NULL) {
      perror("Error opening output file");
      return 0;
    }

    for (int y = 0; y < ny2; y++) {
      for (int x = 0; x < nx2; x++) {
        fprintf(fhdTFile_out, " %E ", Fl_HetT[y * nx2 + x]);
      }
      fprintf(fhdTFile_out, " \n ");
    }

    fclose(fhdTFile_out);
    // Free memory
    free(Fl_HetT);

    time3 = clock();
    printf("Fluorescence simulation time: %.2f sec\n\n",
         (double)(time3 - time1) /CLOCKS_PER_SEC);
  }

  hipProfilerStop();

  //if (Fx != NULL)
  //free(Fx);
  //FreeSimulationStruct(simulations, n_simulations);

  if (fhd_sim_photons == number_phd_photons &&
     (fluor_sim_photons == simulations[0].number_of_photons * fhd_size || simulations[0].do_fl_sim == 0))
     printf("All done, no errors! :)\n");
  else printf ("Simulation finished, some photons were not properly simulated. \n");
  printf("Total time: %.2f sec.\n", (double)(clock() - time0) /CLOCKS_PER_SEC);
  printf("Total simulated photons:\n");
  printf("\t %li FHD photons.\n", fhd_sim_photons);
  printf("\t %li Fluorescence photons.\n", fluor_sim_photons);
  printf("#############################################\n\n");
  return 0;
}
