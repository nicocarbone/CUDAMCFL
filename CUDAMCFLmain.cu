#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////
//
// TODO: CUDAMCFL Description
//
///////////////////////////////////////////////////////////////

/*	This file is part of CUDAMCFL.

    CUDAMCFL is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUDAMCFL is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUDAMCML_INC.  If not, see <http://www.gnu.org/licenses/>.*/

#include "CUDAMCFL.h"
#include "cutil.h"
#include <float.h> //for FLT_MAX
#include <limits.h>
#include <stdio.h>
//#include <string.h>

__device__ __constant__ unsigned long long num_photons_dc[1];
__device__ __constant__ unsigned int n_layers_dc[1];
__device__ __constant__ unsigned int start_weight_dc[1];
__device__ __constant__ LayerStruct layers_dc[MAX_LAYERS];
__device__ __constant__ DetStruct det_dc[1];
__device__ __constant__ IncStruct inclusion_dc[1];
__device__ __constant__ unsigned int ignoreAdetection_dc[1];
__device__ __constant__ unsigned int fhd_activated_dc[1];
__device__ __constant__ float xi_dc[1];
__device__ __constant__ float yi_dc[1];
__device__ __constant__ float zi_dc[1];
__device__ __constant__ float dir_dc[1];
__device__ __constant__ float esp_dc[1];

#include "CUDAMCFLio.cu"
#include "CUDAMCFLmem.cu"
#include "CUDAMCFLrng.cu"
#include "CUDAMCFLtransport.cu"

// wrapper for device code - FHD Simulation
unsigned long long DoOneSimulation(SimulationStruct *simulation, unsigned long long *x,
                     unsigned int *a, double *tempfhd) {
  MemStruct DeviceMem;
  MemStruct HostMem;
  unsigned int threads_active_total = 1;
  unsigned int i, ii;

  // Output matrix size
  const int num_x = (int)(4 * (simulation->esp) * (float)TAM_GRILLA);
  const int num_y = (int)(4 * (simulation->esp) * (float)TAM_GRILLA);
  const int num_z = (int)((simulation->esp) * (float)TAM_GRILLA);
  const int fhd_size = num_x * num_y * num_z;
  //const int fhd_size = num_x + num_x * (num_y + num_z * num_y); //x + HEIGHT* (y + WIDTH* z)

  hipError_t cudastat;
  clock_t time1, time2;

  // Start the clock
  time1 = clock();

  // x and a are already initialised in memory
  HostMem.x = x;
  HostMem.a = a;

  InitMemStructs(&HostMem, &DeviceMem, simulation);

  InitDCMem(simulation);

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(NUM_BLOCKS);

  LaunchPhoton_Global<<<dimGrid, dimBlock>>>(DeviceMem);
  CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
  cudastat = hipGetLastError();           // Check if there was an error
  if (cudastat)
    printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

  i = 0;
  while (threads_active_total > 0) {
    i++;
    // run the kernel
    MCd<<<dimGrid, dimBlock>>>(DeviceMem);
    CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
    cudastat = hipGetLastError();           // Check if there was an error
    if (cudastat)
      printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

    // Copy thread_active from device to host
    CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active,
                              NUM_THREADS * sizeof(unsigned int),
                              hipMemcpyDeviceToHost));
    threads_active_total = 0;
    for (ii = 0; ii < NUM_THREADS; ii++)
      threads_active_total += HostMem.thread_active[ii];

    CUDA_SAFE_CALL(hipMemcpy(HostMem.num_terminated_photons,
                              DeviceMem.num_terminated_photons,
                              sizeof(unsigned long long), hipMemcpyDeviceToHost));
    if (i == 50)
      printf("Estimated FHD simulation time: %.0f secs.\n",
             (double)(clock() - time1) / CLOCKS_PER_SEC *
                 (double)(simulation->number_of_photons /
                          *HostMem.num_terminated_photons));
    if (fmod(i, 200) == 0) printf("."); fflush(stdout);
    if (fmod(i, 10000) == 0)
      printf("\nRun %u, %llu photons simulated\n", i,
             *HostMem.num_terminated_photons);

  }

  CopyDeviceToHostMem(&HostMem, &DeviceMem, simulation);

  time2 = clock();

  printf("\nSimulation time: %.2f sec\n\n",
         (double)(time2 - time1) / CLOCKS_PER_SEC);

  printf("Writing excitation results...\n");
  Write_Simulation_Results(&HostMem, simulation, time2-time1);
  printf("FHD Simulation done!\n");

  unsigned long long photons_finished = *HostMem.num_terminated_photons;

  // Normalize and write output matrix
  for (int xyz = 0; xyz < fhd_size; xyz++) {
    tempfhd[xyz] = ((double)HostMem.fhd[xyz]/(0xFFFFFFFF*photons_finished));//*(double)NUMSTEPS_GPU;//((double)HostMem.fhd[xyz]+(double)FLT_MAX)*10. / photons_finished;
                   //* ((1/(float)TAM_GRILLA) * (1/(float)TAM_GRILLA) * (1/(float)TAM_GRILLA)));
  }

  printf ("Photons simulated: %llu\n\n", photons_finished);
  FreeMemStructs(&HostMem, &DeviceMem);
  return photons_finished;
}

// wrapper for device code - fluorescence Simulation
unsigned long long DoOneSimulationFl(SimulationStruct *simulation, unsigned long long *x,
                       unsigned int *a, unsigned long long *tempvoxel) {
  MemStruct DeviceMem;
  MemStruct HostMem;
  unsigned int threads_active_total = 1;
  unsigned int i, ii;

  // Size of output matrix
  const int nx2 = simulation->det.nx;
  const int ny2 = simulation->det.ny;
  const int xy_size = nx2 + ny2 * nx2;

  hipError_t cudastat;

  // x and a are already initialised in memory
  HostMem.x = x;
  HostMem.a = a;

  InitMemStructs(&HostMem, &DeviceMem, simulation);

  InitDCMem(simulation);

  dim3 dimBlock(NUM_THREADS_PER_BLOCK);
  dim3 dimGrid(NUM_BLOCKS);

  LaunchPhoton_Global<<<dimGrid, dimBlock>>>(DeviceMem);
  CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
  cudastat = hipGetLastError();           // Check if there was an error
  if (cudastat)
    printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

  i = 0;
  int watchdog = 0;
  while (threads_active_total > 0) {
    i++;
    watchdog++;
    // run the kernel
    MCd<<<dimGrid, dimBlock>>>(DeviceMem);
    CUDA_SAFE_CALL(hipDeviceSynchronize()); // Wait for all threads to finish
    cudastat = hipGetLastError();           // Check if there was an error
    if (cudastat)
      printf("Error code=%i, %s.\n", cudastat, hipGetErrorString(cudastat));

    // Copy thread_active from device to host
    CUDA_SAFE_CALL(hipMemcpy(HostMem.thread_active, DeviceMem.thread_active,
                              NUM_THREADS * sizeof(unsigned int),
                              hipMemcpyDeviceToHost));
    threads_active_total = 0;
    for (ii = 0; ii < NUM_THREADS; ii++)
      threads_active_total += HostMem.thread_active[ii];

    CUDA_SAFE_CALL(hipMemcpy(HostMem.num_terminated_photons,
                              DeviceMem.num_terminated_photons,
                              sizeof(unsigned long long), hipMemcpyDeviceToHost));

    if (watchdog > 10000) {
      // If we are still running after 10000 steps, something definetly went wrong.
      printf("\nWARNING: Breaking out of loop...\n");
      return 0;
    }
  }


  CopyDeviceToHostMem(&HostMem, &DeviceMem, simulation);

  for (int ijk = 0; ijk < xy_size; ijk++) {
    // Reflection
    if (simulation->do_fl_sim == 1) tempvoxel[ijk] = HostMem.Rd_xy[ijk];
    // Transmission
    if (simulation->do_fl_sim == 2) tempvoxel[ijk] = HostMem.Tt_xy[ijk];
  }

  unsigned long long photons_finished = *HostMem.num_terminated_photons;
  FreeMemStructs(&HostMem, &DeviceMem);
  return photons_finished;
}

int main(int argc, char *argv[]) {

  clock_t time0 = clock();
  SimulationStruct *simulations;
  int n_simulations;
  unsigned long long seed =
      (unsigned long long)time(NULL); // Default, use time(NULL) as seed
  int ignoreAdetection = 0;
  char *filename;
  char *filenamefl;
  //char *filenamefl3dx;
  //char *filenamefl3dy;
  //char *filenamefl3dz;
  //char *filenamefl3dzInc;
  unsigned long fhd_sim_photons;

  if (argc < 2) {
    printf("Not enough input arguments!\n");
    return 1;
  } else {
    filename = argv[1];
  }

  printf("\nExecuting %s... \n", filename);
  printf("____________________________________________________________________\n\n");

  if (interpret_arg(argc, argv, &seed, &ignoreAdetection))
    return 1;

  n_simulations =
      read_simulation_data(filename, &simulations, ignoreAdetection);

  if (n_simulations == 0) {
    printf("Something wrong with read_simulation_data!\n");
    return 1;
  } else {
    printf("Read %d simulations\n\n", n_simulations);
  }

  printf("Running FHD simulation...\n");

  // Allocate memory for RNG's
  unsigned long long x[NUM_THREADS];
  unsigned int a[NUM_THREADS];

  // Init RNG's
  if (init_RNG(x, a, NUM_THREADS, "safeprimes_base32.txt", seed))
    return 1;

  // Store in local variables the number of voxels in each direction
  const int num_x = (int)(4 * (simulations[0].esp) * (float)TAM_GRILLA);
  const int num_y = (int)(4 * (simulations[0].esp) * (float)TAM_GRILLA);
  const int num_z = (int)((simulations[0].esp) * (float)TAM_GRILLA);
  //const int fhd_size = num_x + num_x * (num_y + num_y * num_z); //x + HEIGHT* (y + WIDTH* z)
  const int fhd_size = num_x * num_y * num_z; //x + HEIGHT* (y + WIDTH* z)

  // FHD simulation
  // Perform all the simulations TODO
  // for(i=0;i<n_simulations;i++)
  //{
  // Run a simulation
  double *Fx;
  Fx = (double *)malloc((fhd_size) * sizeof(double));
  fhd_sim_photons = DoOneSimulation(&simulations[0], x, a, Fx);
  //}



  // Outputting FHD files for debug
  printf("Writing FHD files...\n"); // TODO

  // ASCII file
  FILE *fhd3DaFile_out;
  char filenamefl3da[STR_LEN];
	for (int ic=0; ic<STR_LEN; ic++) filenamefl3da[ic] = simulations[0].outp_filename[ic];
  strcat(filenamefl3da, "_FHD-Ascii.dat");

  fhd3DaFile_out = fopen(filenamefl3da, "w");
  if (fhd3DaFile_out == NULL) {
    perror("Error opening output file");
    return 0;
  }

  fprintf(fhd3DaFile_out, "%llu\t%llu\t%llu\n", num_x,num_y,num_z);

  for (int xyz = 0; xyz < fhd_size; xyz++) {
    fprintf(fhd3DaFile_out, "%.10E\n", Fx[xyz]);
  }

  fclose(fhd3DaFile_out);

  /*
  // Binary file
  FILE *fhd3DbFile_out;
  char filenamefl3db[STR_LEN];
	for (int ic=0; ic<STR_LEN; ic++) filenamefl3db[ic] = simulations[0].outp_filename[ic];
  strcat(filenamefl3db, "_FHD-Binary.dat");

  fhd3DbFile_out = fopen(filenamefl3db, "wb");
  if (fhd3DbFile_out == NULL) {
    perror("Error opening output file");
    return 0;
  }

  */

  // Fluorescence simulation

  // Initialize GPU RNG
  seed = (unsigned long long)time(NULL); // Default, use time(NULL) as seed
  if (init_RNG(x, a, NUM_THREADS, "safeprimes_base32.txt", seed))
    return 1;

  unsigned long fluor_sim_photons = 0; //Nro of simulated fluorescence photons

  if (simulations[0].do_fl_sim != 0){
    printf("Flourescence simulation... \n");
    int count_failed = 0;
    // Store in local variables the number of pixels and calculate image size
    const int nx2 = simulations[0].det.nx;
    const int ny2 = simulations[0].det.ny;
    const int xy_size = nx2 + ny2 * nx2;

    // Pixel size for Normalization
    const double dx = simulations[0].det.dx;
    const double dy = simulations[0].det.dy;

    // Initialize arrays
    double *Fl_Het;          // Final fluorescence image
    long voxel_finished = 0; // Nro of voxel simulated
    long voxel_inside = 0;   // Nro of voxel simulated inside inclusion
    long voxel_outside = 0;  // Nro of voxel simulated outside inclusion
    //const long for_size =
        num_x * num_y * num_z; // Total number of voxels to be simulated
    float xi, yi, zi;          // Temporal variable to store the voxel coordinates
    double voxelw; // Temporal variable to store the voxel scale factor
    clock_t time1,
        time2, time3; // Variable to store the timestamps used for run time stimation

    // Allocate and initialize to zero image matrix
    Fl_Het = (double *)malloc(xy_size * sizeof(double));
    for (int ijk = 0; ijk < xy_size; ijk++) {
      Fl_Het[ijk] = 0.0;
    }

    // Simulations parameters
    for (int n = 0; n < simulations[0].n_layers + 2;
        n++) { // Set mua to fluorescence value for every layer
        simulations[0].layers[n].mua = simulations[0].layers[n].muaf;
    }
    simulations[0].number_of_photons = (unsigned long long)simulations[0].number_of_photons_per_voxel; // Number of photons per voxel
    simulations[0].dir = 0.0f;        // Isotropic source
    simulations[0].fhd_activated = 0; // Don't accumulate fhd

    printf("Total fotons to be simulated: %lli over %li voxels\n",
          simulations[0].number_of_photons * fhd_size, fhd_size);

    // Loop through the voxels
    for (int ix = 0; ix < num_x; ix++) {
      for (int iy = 0; iy < num_y; iy++) {
        for (int iz = 0; iz < num_z; iz++) {
          if (ix == 0 && iy == 0 && iz == 0)
            time1 = clock(); // For the first xyz voxel, take first timestamp

          // Set source position
          xi = (ix / (float)TAM_GRILLA) - 2* simulations[0].esp;
          yi = (iy / (float)TAM_GRILLA) - 2* simulations[0].esp;
          zi = (iz / (float)TAM_GRILLA);

          simulations[0].xi = xi;
          simulations[0].yi = yi;
          simulations[0].zi = zi;

          // Locate layer of voxel (we need it to retrieve apropiate albedo)
          int found = 0;
          int nl = 1;
          while (nl < simulations[0].n_layers + 2 && found != 1) {
            if (zi < simulations[0].layers[nl].z_max &&
                zi >= simulations[0].layers[nl].z_min) {
              found = 1;
            } else
              nl++;
          }

          // Do the voxel simulation
          unsigned long long *tempret;
          tempret =
              (unsigned long long *)malloc(xy_size * sizeof(unsigned long long));
          unsigned long long voxel_status = DoOneSimulationFl(&simulations[0], x, a, tempret);

          // Check if inside inclusion and calculate scale value accordingly
          if ((xi - simulations[0].inclusion.x) *
                      (xi - simulations[0].inclusion.x) +
                  (yi - simulations[0].inclusion.y) *
                      (yi - simulations[0].inclusion.y) +
                  (zi - simulations[0].inclusion.z) *
                      (zi - simulations[0].inclusion.z) <
                simulations[0].inclusion.r * simulations[0].inclusion.r) {
          // voxel inside inclusion
            voxelw = ((double)simulations[0].inclusion.eY *
                    (double)(1 - simulations[0].inclusion.albedof) *
                    Fx[ix + num_x * (iy + iz * num_y)]) /
                    (double)(voxel_status * 0xFFFFFFFF);
            voxel_inside++;
          } else {
            // voxel ouside inclusion
            voxelw = ((double)simulations[0].layers[nl].eY *
                    (double)(1 - simulations[0].layers[nl].albedof) *
                    Fx[ix + num_x * (iy + iz * num_y)]) /
                    (double)(voxel_status * 0xFFFFFFFF);
            voxel_outside++;
          }


          if (voxel_status == 0) {
            printf("Voxel %f, %f, %f failed.\n", xi,yi,zi);
            count_failed += 1;
            }

          fluor_sim_photons += voxel_status;

          // Accumulate image
          for (int ij = 0; ij < xy_size; ij++) {
            double tempvw = voxelw * (double)tempret[ij];
            //printf ("%E\n", tempvw);
            if (Fl_Het[ij] + tempvw < DBL_MAX) Fl_Het[ij] += tempvw/(dx*dy);
					}
          voxel_finished++;
          free(tempret);
          if (fmod(voxel_finished, 200) == 0) printf("."); fflush(stdout);
          if (fmod(voxel_finished, 10000) == 0)
            printf("\n%li of %li voxels finished\n", voxel_finished, fhd_size);
          if (voxel_finished == 99) { // Second timestamp after 99 voxels run (so
                                    // it displays before the first progression
                                    // report)
          time2 = clock();
            printf("Estimated fluorescence simulation time: %.0f sec\n\n",
                 (double)(time2 - time1) * fhd_size / CLOCKS_PER_SEC / 99);
          }
        }
      }
    }

    printf("\n\nFlourescence simulation finished!\n");
    printf("Voxels inside inclusion: %li\n", voxel_inside);
    printf("Voxels outside inclusion: %li\n", voxel_outside);
    printf("Voxels failed: %i\n", count_failed);

    printf("Writing results file...\n"); // TODO
    FILE *fhdFile_out;
    filenamefl = simulations[0].outp_filename;
    strcat(filenamefl, "_Fl.dat");

    fhdFile_out = fopen(filenamefl, "w");
    if (fhdFile_out == NULL) {
      perror("Error opening output file");
      return 0;
    }

    for (int y = 0; y < ny2; y++) {
      for (int x = 0; x < nx2; x++) {
        fprintf(fhdFile_out, " %E ", Fl_Het[y * nx2 + x]);
      }
      fprintf(fhdFile_out, " \n ");
    }

    fclose(fhdFile_out);
    // Free memory
    free(Fl_Het);
    time3 = clock();
    printf("Fluorescence simulation time: %.2f sec\n\n",
         (double)(time3 - time1) /CLOCKS_PER_SEC);
  }
  free(Fx);
  FreeSimulationStruct(simulations, n_simulations);

  printf("All done! :)\n");
  printf("Total time: %.2f sec.\n", (double)(clock() - time0) /CLOCKS_PER_SEC);
  printf("Total simulated photons:\n");
  printf("\t %li FHD photons.\n", fhd_sim_photons);
  printf("\t %li Fluorescence photons.\n", fluor_sim_photons);
  printf("#############################################\n\n");
  return 0;
}
